#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void initA(int *A, int numElements) {
	 i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i < numElements && j<numElements)
		if(i==j)
			A[i*numElements+j]=1;
		else A[i*numElements+j]=0;
}

__global__ void initB(int *B, int numElements) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < numElements)
		B[i]=i;
}


__global__ void MatAdd(int *A, int *B, int *C, int numElements){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int sum=0;
	if (i < numElements ){
		for(int j=0;j<numElements;j++)
			sum+=A[j*numElements+i]*B[j];
	}
	C[i]=sum;
}



__host__ int main(void) {
	hipError_t err = hipSuccess;
	hipEvent_t start, stop;
	float gpuTime=0.0f;
	const int numElements = 21504;
	const int k=16;
	size_t size1 = numElements * numElements * sizeof(int);
	size_t size2 = numElements * sizeof(int);

	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	int *h_C = (int *)malloc(size2);
	for(int i=0;i<numElements;i++)h_C[i]=0;
	
	int *d_A = NULL;
	err = hipMalloc((void **)&d_A, size1);
	if (err != hipSuccess){
		printf("malloc_d_A error\n");
		return 0;
	}
	
	int *d_B = NULL;
	err = hipMalloc((void **)&d_B, size2);
	if (err != hipSuccess){
		printf("malloc_d_B error\n");
		return 0;
	}

	int *d_C= NULL;
	err = hipMalloc((void **)&d_C,size2);
	if (err != hipSuccess){
		printf("malloc_d_C error\n");
		return 0;
	}
	
	dim3 threadsPerBlock(k,k);
	
	int BlockSizeI=((numElements + k -1)/k);
                                              
	int BlockSizeC=((numElements + k*k -1)/(k*k));

	dim3 blocksPerGrid (BlockSizeI, BlockSizeI);
	
	initA<<<blocksPerGrid, threadsPerBlock>>>(d_A, numElements);
	err = hipGetLastError();
	if (err != hipSuccess){
		printf("initA error\n");
		return 0;
	}

	threadsPerBlock = k*k;
	blocksPerGrid = BlockSizeC;
	
	initB<<<blocksPerGrid, threadsPerBlock>>>(d_B, numElements);
	err = hipGetLastError();
	if (err != hipSuccess){
		printf("initB error\n");
		return 0;
	}
		
	hipEventRecord(start,0);

	
	MatAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
	err = hipGetLastError();
	if (err != hipSuccess){
		printf("MatAdd error\n");
		return 0;
	}
	
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime,start,stop);
	
	printf("Time: %.9f msec.",gpuTime);
	
	hipMemcpy(h_C, d_C, size2, hipMemcpyDeviceToHost);
	
	err = hipFree(d_A);
	err = hipFree(d_B);
	err = hipFree(d_C);
	
	printf("\n");
	for(int i=0; i<numElements;i++)printf("%d ",h_C[i]);
	printf("\n");
	
	free(h_C);
	
	return 0;
}
