
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE  5          // submatrix size 
__global__ void matMult ( int * a, int * b, int n, int * c )
{
    int bx = blockIdx.x;        // block index
    int by = blockIdx.y;

    int tx = threadIdx.x;       // thread index
    int ty = threadIdx.y;
    
                                // Index of the first sub-matrix of A processed by the block
    int aBegin = n * BLOCK_SIZE * by;
    int aEnd = aBegin + n - 1;
                                // Step size used to iterate through the sub-matrices of A
    int aStep = BLOCK_SIZE;
                                // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;
                                // Step size used to iterate through the sub-matrices of B
    int bStep = BLOCK_SIZE * n;
    int sum = 0;           // computed subelement
    
    for ( int ia = aBegin, ib = bBegin; ia <= aEnd; ia += aStep, ib += bStep )
    {
                            // Shared memory for the sub-matrix of A
        __shared__ int as [BLOCK_SIZE][BLOCK_SIZE];
                            // Shared memory for the sub-matrix of B
        __shared__ int bs [BLOCK_SIZE][BLOCK_SIZE];
        
                            // Load the matrices from global memory to shared memory;
        as [ty][tx] = a [ia + n * ty + tx];
        bs [ty][tx] = b [ib + n * ty + tx];
        
        __syncthreads();    // Synchronize to make sure the matrices are loaded
        
                            // Multiply the two matrices together;
        for ( int k = 0; k < BLOCK_SIZE; k++ )
            sum += as [ty][k] * bs [k][tx];
            
                            // Synchronize to make sure that the preceding
                            // computation is done before loading two new
                            // sub-matrices of A and B in the next iteration
        __syncthreads();
    }
    
                            // Write the block sub-matrix to global memory;
                            // each thread writes one element
    int ic = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    
    c [ic + n * ty + tx] = sum;
}

int main ( int argc, char *  argv [] )
{
    int N=10;
    int numBytes = N * N * sizeof ( int );

                    // allocate host memory
    int * a = new int [N*N];
    int * b = new int [N*N];
    int * c = new int [N*N];
    
    for ( int i = 0; i < N; i++ )
        for ( int j = 0; j < N; j++ )
        {
            a [N*i+j] = 1;
            b [N*i+j] = 1;
        }
        
                    // allocate device memory
    int * adev = NULL;
    int * bdev = NULL;
    int * cdev = NULL;
    
    hipMalloc ( (void**)&adev, numBytes );
    hipMalloc ( (void**)&bdev, numBytes );
    hipMalloc ( (void**)&cdev, numBytes );

                    // set kernel launch configuration
    dim3 threads ( BLOCK_SIZE, BLOCK_SIZE );
    dim3 blocks  ( N / threads.x, N / threads.y);

                    // create cuda event handles
    hipEvent_t start, stop;
    float gpuTime = 0.0f;

    hipEventCreate ( &start );
    hipEventCreate ( &stop );
    
                    // asynchronously issue work to the GPU (all to stream 0)
    hipEventRecord ( start, 0 );
    hipMemcpy      ( adev, a, numBytes, hipMemcpyHostToDevice );
    hipMemcpy      ( bdev, b, numBytes, hipMemcpyHostToDevice );
    
    matMult<<<blocks, threads>>> ( adev, bdev, N, cdev );
    
    hipMemcpy      ( c, cdev, numBytes, hipMemcpyDeviceToHost );
    hipEventRecord ( stop, 0 );

    hipEventSynchronize ( stop );
    hipEventElapsedTime ( &gpuTime, start, stop );
 
	for(int i=0;i<N;i++)
		{
		for(int j=0;j<N;j++)
			printf("%d ",c[i]);			

	printf("\n");
	}
                        // print the cpu and gpu times
    printf("time spent executing by the GPU: %.2f millseconds\n", gpuTime );
   
                    // release resources
    hipEventDestroy ( start );
    hipEventDestroy ( stop  );
    hipFree         ( adev  );
    hipFree         ( bdev  );
    hipFree         ( cdev  );

    delete a;
    delete b;
    delete c;

    return 0;
}
